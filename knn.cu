#include "hip/hip_runtime.h"
#include "knn.cuh"

__global__
void add(int n, float* x, float* y)
{
	for (int i = 0; i < n; i++)
		y[i] = x[i] + y[i];
}

__global__ void init(int n, float* x, float* y) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
}

KNNClassifier::KNNClassifier(std::vector<std::string>& fileNames, int resolution)
{
	
	// ����� �������� resolution � trainDataSize, ��� ��� ������ �����������
	this->resolution = resolution;
	this->trainDataSize = fileNames.size();

	// �������� ������ ��� �������� ������
	hipError_t rsp;
	rsp = hipMalloc(&this->trainDataPtr, sizeof(uint8_t) * resolution * resolution * this->trainDataSize);
	if (rsp != hipError_t::hipSuccess) {
		throw std::exception("Could not allocate memory for training samples: " + rsp);
	}
	// �������� ������ ��� �������������� �������� ������
	rsp = hipMalloc(&this->trainClsPtr, sizeof(char) * this->trainDataSize);
	if (rsp != hipError_t::hipSuccess) {
		throw std::exception("Could not allocate memory for training classifiers: " + rsp);
	}
	
	const int dataChunkSize = sizeof(uint8_t) * resolution * resolution;
	for (int idx = 0; idx < this->trainDataSize; idx++) {

		// ��������� � ����� � �������� (1�8U)
		cv::Mat mat = cv::imread(fileNames[idx], cv::ImreadModes::IMREAD_GRAYSCALE);
		if (mat.empty()) {
			std::stringstream errMsgStream;
			errMsgStream << "Error reading file " << fileNames[idx] << ". Mat is empty.";
			throw std::exception(errMsgStream.str().c_str());
		}
		else if (mat.size().height != resolution || mat.size().width != resolution) {
			std::stringstream errMsgStream;
			errMsgStream << "Error reading file " << fileNames[idx] << ". Training data image has wrong resolution: " << 
				mat.size().width << "x" << mat.size().height;
			throw std::exception(errMsgStream.str().c_str());
		}

		rsp = hipMemcpy(this->trainDataPtr + idx * dataChunkSize, &mat, dataChunkSize, hipMemcpyKind::hipMemcpyHostToDevice);
		CHECK_CUDA(rsp, true, "Cannot load file ", fileNames[idx]);

		// ���� ��� �� � �� �������� ��������, ������� �� ��� � �����. ����������
		// �� *nix ��� ������� �������� �� �����, ����� ���������
		const char cls = fileNames[idx][fileNames[idx].find_last_of("//") + 1];
		rsp = hipMemset(this->trainClsPtr + idx, cls, 1);
		CHECK_CUDA(rsp, true, "Cannot save classifier ", cls, " for file ", fileNames[idx]);

	}

	/*
	char* testSample = (char*) malloc(17 * sizeof(char));
	memset(testSample, 0, 17 * sizeof(char));
	rsp = hipMemcpy(testSample, this->trainClsPtr, 16 * sizeof(char), hipMemcpyKind::hipMemcpyDeviceToHost);
	testSample[16] = '\0';
	std::cout << "16 first classifiers from GPU memory: " << testSample << std::endl;
	*/

	std::cout << this->trainDataSize << " training samples successfully loaded!" << std::endl <<
		"KNNClassifier " << this << " has been successfully initialized." << std::endl;

}

std::vector<CharacterClassification> KNNClassifier::classifyCharacters(std::vector<cv::Mat>& chars)
{
	// ������ CUDA Streams �� ���������� ���� �� �������������
	// ����� ���������� ���������� �����
	hipError_t rsp;
	std::vector<hipStream_t*> streams;

	for (cv::Mat& mat : chars) {
		
		// ������� �����
		hipStream_t* stream;
		rsp = hipStreamCreate(stream);
		CHECK_CUDA(rsp, true, "Cannot initialize CUDA stream.");
		
		// �������� � �������� ����� ������� ����� �������� �� GPU
		uint8_t* requestedMatPtr;
		rsp = hipMallocAsync(&requestedMatPtr, this->dataChunkSize, *stream);
		CHECK_CUDA(rsp, true, "Could not allocate memory for supplied image");
		rsp = hipMemcpyAsync(requestedMatPtr, &mat, this->dataChunkSize, hipMemcpyKind::hipMemcpyHostToDevice, *stream);
		CHECK_CUDA(rsp, true, "Could not transfer data of the supplied image to the GPU");

		// ���� ��� ��, �������������� ������, � ������� ����� ��������� ����������
		// ����� �������������� ���������� � �������. �������� ��� �������� �������
		uint32_t* distPtr;
		rsp = hipMallocAsync(&distPtr, this->trainDataSize * sizeof(uint32_t), *stream);
		CHECK_CUDA(rsp, true, "Could not allocate memory for neighbor distances");
		rsp = hipMemsetAsync(distPtr, UINT_MAX, this->trainDataSize, *stream);
		CHECK_CUDA(rsp, true, "Could not initialize distances array");

		// �� ������ ���������� ������ � ����� ����� ����������

	}

	return std::vector<CharacterClassification>();

}

KNNClassifier::~KNNClassifier()
{

	hipError_t rsp;

	rsp = hipFree(this->trainDataPtr);
	CHECK_CUDA(rsp, true, "Could not free train data memory on GPU...");
	
	rsp = hipFree(this->trainClsPtr);
	CHECK_CUDA(rsp, true, "Could not free train classifiers memory on GPU...");

}

int tutorial(void) {

	int N = 1 << 20;

	float* x, * y;
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// initialize x and y arrays on the host
	add<<<1, 1 >>> (N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;

}