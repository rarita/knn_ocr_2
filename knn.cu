#include "hip/hip_runtime.h"
#include "knn.cuh"

__global__
void add(int n, float* x, float* y)
{
	for (int i = 0; i < n; i++)
		y[i] = x[i] + y[i];
}

__global__ void init(int n, float* x, float* y) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
}

__global__ void dist(uint8_t* trainPtr, uint8_t* inputPtr, uint32_t* distPtr, int res, int tds) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int resSq = res * res;
	for (int idx = index; idx < resSq * tds; idx += stride) {
		// XOR-�� �����. ������� ������������� �������� � �������� �� ���� ��������
		// ��� �������. �������� ������� 0xFF
		uint8_t xorEd = trainPtr[idx] ^ inputPtr[idx % resSq];
		#ifndef DBG_CUDA_KERNEL
		printf("dist: index:%d; idx: %d; stride: %d; train data at idx: %d; input at idx: %d; xor at idx: %d; char: %d\n", 
			index, idx, stride, trainPtr[idx], inputPtr[idx % resSq], xorEd, idx / resSq
		);
		#endif
		// ��������� � ������ ����������
		// ��������� ��� ��� race condition :( ��� �� �� ������� �� ����
		distPtr[idx] = xorEd / 255; // �� 0 �� 1
	}
}

__global__ void dist_reduce(uint32_t* distPtr, uint32_t* distRedPtr, int res, int tds) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int resSq = res * res;
	if (index > tds) {
		return;
	}

	#pragma unroll
	for (int idxRes = 0; idxRes < resSq; idxRes++) {
		#ifdef DBG_CUDA_KERNEL
		printf("dist_reduce: index: %d; idx: %d; idxRes: %d\n", index, idxRes);
		#endif
		distRedPtr[index] += distPtr[(index * resSq) + idxRes];
	}

}

KNNClassifier::KNNClassifier(std::vector<std::string>& fileNames, int resolution)
{
	
	// ����� ������ �������������
	const std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

	// ����� �������� � �������� ��� ��������� ���������, ��� ��� ������ �����������
	this->resolution = resolution;
	this->trainDataSize = fileNames.size();
	this->dataChunkSize = resolution * resolution * sizeof(uint8_t);

	// �������� ������ ��� �������� ������
	hipError_t rsp;
	rsp = hipMalloc(&this->trainDataPtr, sizeof(uint8_t) * resolution * resolution * this->trainDataSize);
	if (rsp != hipError_t::hipSuccess) {
		throw std::exception("Could not allocate memory for training samples: " + rsp);
	}
	// �������� ������ ��� �������������� �������� ������
	rsp = hipMalloc(&this->trainClsPtr, sizeof(char) * this->trainDataSize);
	if (rsp != hipError_t::hipSuccess) {
		throw std::exception("Could not allocate memory for training classifiers: " + rsp);
	}

	#pragma omp parallel for
	for (int idx = 0; idx < this->trainDataSize; idx++) {

		// ��������� � ����� � �������� (1�8U)
		cv::Mat mat = cv::imread(fileNames[idx], cv::ImreadModes::IMREAD_GRAYSCALE);

		// ��������� ��� ������� � ��� �� � ������������
		if (mat.empty()) {
			std::stringstream errMsgStream;
			errMsgStream << "Error reading file " << fileNames[idx] << ". Mat is empty.";
			throw std::exception(errMsgStream.str().c_str());
		}
		else if (mat.size().height != resolution || mat.size().width != resolution) {
			std::stringstream errMsgStream;
			errMsgStream << "Error reading file " << fileNames[idx] << ". Training data image has wrong resolution: " << 
				mat.size().width << "x" << mat.size().height;
			throw std::exception(errMsgStream.str().c_str());
		}

		// ��������� �� ������ ������
		cv::threshold(mat, mat, 178, 255, cv::ThresholdTypes::THRESH_BINARY);

		cv::Mat flat = mat.reshape(1, mat.total() * mat.channels());
		std::vector<uchar> vec = mat.isContinuous() ? flat : flat.clone();

		rsp = hipMemcpyAsync(this->trainDataPtr + idx * this->dataChunkSize, vec.data(), this->dataChunkSize, hipMemcpyKind::hipMemcpyHostToDevice);
		CHECK_CUDA(rsp, true, "Cannot load file ", fileNames[idx]);

		// ���� ��� �� � �� �������� ��������, ������� �� ��� � �����. ����������
		// �� *nix ��� ������� �������� �� �����, ����� ���������
		const char cls = fileNames[idx][fileNames[idx].find_last_of("//") + 1];
		rsp = hipMemset(this->trainClsPtr + idx, cls, 1);
		CHECK_CUDA(rsp, true, "Cannot save classifier ", cls, " for file ", fileNames[idx]);

	}

	rsp = hipDeviceSynchronize();
	CHECK_CUDA(rsp, true, "Could not synchronize CUDA device after uploading data");

	/*
	char* testSample = (char*) malloc(17 * sizeof(char));
	memset(testSample, 0, 17 * sizeof(char));
	rsp = hipMemcpy(testSample, this->trainClsPtr, 16 * sizeof(char), hipMemcpyKind::hipMemcpyDeviceToHost);
	testSample[16] = '\0';
	std::cout << "16 first classifiers from GPU memory: " << testSample << std::endl;
	*/


	const std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	std::cout << this->trainDataSize << " training samples successfully loaded in " <<
		std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count() << " ms. " <<
		"KNNClassifier " << this << " has been successfully initialized." << std::endl;

}

void uploadMatToDev(thrust::device_vector<uint8_t>& dVec, int offset, cv::Mat& mat) {
	// ���������, ��� �������� ����� ������ ���
	assert(mat.type() == CV_8UC1);
	cv::Mat flat = mat.reshape(1, mat.total() * mat.channels());
	std::vector<uchar> vec = mat.isContinuous() ? flat : flat.clone();
	thrust::host_vector<uint8_t> requestedMatHVec(vec);
	thrust::copy(requestedMatHVec.begin(), requestedMatHVec.end(), dVec.begin() + offset);
}

void printDeviceTexture(thrust::device_vector<uint8_t>& dVec, int res, int offset) {
	thrust::host_vector<uint8_t> matsHVec(dVec);
	printf("Image in device memory (8 bit single channel):\n");
	for (int y = 0; y < res; y++) {
		for (int x = 0; x < res; x++) {
			printf("%03d|", matsHVec[y * res + x + (offset * res * res)]);
		}
		printf("\n");
	}
	printf("\n");
}

template<typename T>
void printDeviceVector(thrust::device_vector<T> dVec) {
	thrust::host_vector<T> hVec(dVec);
	printf("Vector size (%d) : ", hVec.size());
	for (T& elem : hVec) {
		printf("%d|", elem);
	}
	printf("\n");
}

void checkCudaAsyncMemMgmtSupport() {
	int attr;
	hipError_t rsp;
	rsp = hipDeviceGetAttribute(&attr, hipDeviceAttributeMemoryPoolsSupported, 0);
	CHECK_CUDA(rsp, true, "Can't probe for memory pools support");
	printf("Device supports memory pooling (async memset): %d\n", attr);
}

std::vector<CharacterClassification> KNNClassifier::classifyCharacters(std::vector<cv::Mat>& chars, int k)
{
	// ������ CUDA Streams �� ���������� ���� �� �������������
	// ����� ���������� ���������� �����
	hipError_t rsp;
	std::vector<hipStream_t> streams;
	std::vector<CharacterClassification> result(chars.size());
	
	const int texSize = this->resolution * this->resolution;

	// ��������� ���������� � ������ GPU ��� chars
	thrust::device_vector<uint8_t> matsDVec(chars.size() * texSize);
	#pragma omp parallel for
	for (int idx = 0; idx < chars.size(); idx++) {
		// ����� ������ ������������ streams, �������� �� ������������ �����. ���������� �������
		uploadMatToDev(matsDVec, idx * texSize, chars[idx]);
	}

	printDeviceTexture(matsDVec, this->resolution, 0);

	for (cv::Mat& mat : chars) {
		
		// ������� ���������� �����
		hipStream_t stream;
		rsp = hipStreamCreate(&stream);
		CHECK_CUDA(rsp, true, "Could not create stream");

		streams.push_back(stream);
		printf("Stream is: %d\n", stream);
		// �������� � �������� ����� ������� ����� �������� � ������ GPU
		thrust::device_vector<uint8_t> requestedMatDVec(this->resolution*this->resolution);
		cv::Mat flat = mat.reshape(1, mat.total() * mat.channels());
		std::vector<uchar> vec = mat.isContinuous() ? flat : flat.clone();
		thrust::host_vector<uint8_t> requestedMatHVec(vec);
		thrust::copy(requestedMatHVec.begin(), requestedMatHVec.end(), requestedMatDVec.begin());

		// ���� ��� ��, �������������� ������, � ������� ����� ��������� ����������
		// ����� �������������� ���������� � �������. �������� ��� �������� �������
		thrust::device_vector<uint32_t> distVec(this->trainDataSize * this->resolution * this->resolution);
		// rsp = hipMallocAsync(&distPtr, this->trainDataSize * sizeof(uint32_t), stream);
		// CHECK_CUDA(rsp, true, "Could not allocate memory for neighbor distances");
		// rsp = hipMemsetAsync(distPtr, UINT_MAX, this->trainDataSize, stream);
		// CHECK_CUDA(rsp, true, "Could not initialize distances array");
		// ��� ���������� ������ � ����������������, ����� ��� ������������ in-place � ������� thrust
		// ������, �������� ����� ������� ���������������
		thrust::device_vector<char> clsCopyVec(this->trainDataSize);
		// thrust-��������� �� ������������ ������ ���������������
		thrust::device_ptr<char> clsDevPtr(this->trainClsPtr);
		thrust::copy(thrust::cuda::par.on(stream), clsDevPtr, clsDevPtr + this->trainDataSize, clsCopyVec.begin());
		// thrust::copy(thrust::cuda::par(*stream), this->trainClsPtr, this->trainClsPtr + sizeof(uint8_t) * this->trainDataSize, clsVec.begin());

		// ��������� �������, ������� ��������� ��� ���������� �����
		// ������� ������������ � ������ �������������� �������.
		// ������ ������� ��� �� �������� thrust::transform � ������������� thrust::bitwise_xor

		uint8_t* reqMatRawPtr = thrust::raw_pointer_cast(requestedMatDVec.data());
		uint32_t* distRawPtr = thrust::raw_pointer_cast(distVec.data());
		
		const int threadsPerBlock = 256; // ���������� (?) ��� ���� ����� (MX130)

		dist<<<4, threadsPerBlock, 0, stream>>> (this->trainDataPtr, reqMatRawPtr, distRawPtr, this->resolution, this->trainDataSize);
		rsp = hipGetLastError();
		CHECK_CUDA(rsp, true, "Kernel launch was unsuccessful", "dist kernel");

		// ��������� ���������� XOR ��� ������ �������� ����������
		thrust::device_vector<uint32_t> distRedVec(this->trainDataSize);
		uint32_t* distRedVecRawPtr = thrust::raw_pointer_cast(distRedVec.data());

		const int blockCount = (this->trainDataSize / threadsPerBlock) + 1;
		dist_reduce <<<blockCount, threadsPerBlock, 0, stream >>> (distRawPtr, distRedVecRawPtr, this->resolution, this->trainDataSize);
		rsp = hipGetLastError();
		CHECK_CUDA(rsp, true, "Kernel launch was unsuccessful", "dist_reduce kernel");

		hipDeviceSynchronize();
		printDeviceVector(distRedVec);

		// ���� ������� TOP-K ������� �������������� �����������
		// thrust::device_ptr<uint32_t> keysVecPtr = thrust::device_pointer_cast<uint32_t>(distPtr);
		// thrust::sort_by_key(thrust::cuda::par(*stream), keysVecPtr, keysVecPtr + this->trainDataSize, clsVec.begin());
		
		// ���� ������������� ����� ����� TOP-K �������
		// ��� ����� ������� ������ �������� ����� reduce_by_key ��� ������� ����� ������ �������
		// thrust::device_ptr<uint8_t> clsVecPtr = thrust::device_pointer_cast(clsVec.data());
		// �������� ����������� constant_iterator<uint8_t>
		// thrust::device_vector<uint8_t> onesVec(k, 1);
		// thrust::device_vector<uint8_t> clsOut(k);
		// thrust::device_vector<uint8_t> cntOut(k);
		// thrust::reduce_by_key(thrust::cuda::par(*stream), clsVecPtr, clsVecPtr + k * sizeof(uint8_t), onesVec.begin(), clsOut.begin(), cntOut.begin());
		// ���� ���-1 � ������������� ������ �� ��������� 
		// thrust::device_ptr<uint8_t> cntOutPtr = thrust::device_pointer_cast<uint8_t>(onesVec.data());
		// thrust::sort_by_key(thrust::cuda::par(*stream), cntOut.begin(), cntOut.end(), clsOut.begin());
		
		// ��������� ������������� ����� � ������� ������� ������ �� ����� �� ����
		// thrust::host_vector<uint8_t> clsHostVec(clsVec);
		// CharacterClassification cc;
		// cc.cls = static_cast<char>(clsHostVec[0]);
		// cc.loc = &mat;
		// result.push_back(cc);

		// �� �������� ���������� ������ � ����� ����� ����������
		// rsp = hipFreeAsync(requestedMatPtr, stream);
		// CHECK_CUDA(rsp, true, "Could not free memory for input texture");
		// rsp = hipFreeAsync(distPtr, stream);
		// CHECK_CUDA(rsp, true, "Could not allocate memory for input neighbours distances");
		rsp = hipStreamDestroy(stream); // ����� ����, �� ������!!!!!!!!!!!!!!!!!!
		CHECK_CUDA(rsp, true, "Could not shut down CUDA stream", stream);

	}

	return result;

}

KNNClassifier::~KNNClassifier(){

	hipError_t rsp;

	rsp = hipFree(this->trainDataPtr);
	CHECK_CUDA(rsp, true, "Could not free train data memory on GPU...");
	
	rsp = hipFree(this->trainClsPtr);
	CHECK_CUDA(rsp, true, "Could not free train classifiers memory on GPU...");

}

void testOMP() {

	printf("Availiable OMP threads system-wide : %d\n", omp_get_max_threads());
	# pragma omp parallel for
	for (int idx = 0; idx < 10; idx++) {
		assert(omp_get_num_threads() > 1);
	}

}

int tutorial(void) {

	int N = 1 << 20;

	float* x, * y;
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// initialize x and y arrays on the host
	add<<<1, 1 >>> (N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;

}